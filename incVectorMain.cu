#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void vAdd(int* A, int num_elements){

	//Posicion del thread
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	printf("Hola desde el hilo %d, en el bloque %d y el hilo %d\n", i, blockIdx.x, threadIdx.x);

	if(i < num_elements){
		A[i] = A[i] + 1;
	}


}



void fError(hipError_t err){
	if(err != hipSuccess){
		printf("Ha ocurrido un error con codigo: %s\n", hipGetErrorString(err));
	}
}


int main(){

	int num_elements = 100000;

	//Reservar espacio en memoria HOST


	int * h_A = (int*)malloc(num_elements * sizeof(int));


	if(h_A == NULL ){
		printf("Error al reservar memoria para los vectores HOST");
		exit(1);
	}



	//Inicializar elementos de los vectores
	for(int i=0; i<num_elements; i++){
		h_A[i] = 10;

	}

	hipError_t err;

	int size = num_elements * sizeof(int);

	int * d_A = NULL;
	err = hipMalloc((void **)&d_A, size);
	fError(err);


	//Copiamos a GPU DEVICE
	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);


	int HilosPorBloque = 256;
	int BloquesPorGrid = (num_elements + HilosPorBloque -1) / HilosPorBloque;


	hipError_t Err;

	//Lanzamos el kernel y medimos tiempos
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	vAdd<<<BloquesPorGrid, HilosPorBloque>>>(d_A, num_elements);
	Err = hipGetLastError();
	fError(Err);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float tiempo_reserva_host;
	hipEventElapsedTime(&tiempo_reserva_host, start, stop);


	printf("Tiempo de suma vectores DEVICE: %f\n", tiempo_reserva_host);

	hipEventDestroy(start);
	hipEventDestroy(stop);


	//Copiamos a CPU el vector C
	err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);


	/*for(int i=0; i<num_elements; i++){
		printf("%i", h_A[i]);
		printf("\n");
	}*/

}







